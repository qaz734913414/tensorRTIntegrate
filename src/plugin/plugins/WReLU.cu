#include "hip/hip_runtime.h"


#include "WReLU.hpp"

typedef TRTInfer::halfloat halfloat;

template<typename _T>
__global__ void relu(_T* input, _T* output, int edge);

template<>
__global__ void relu(float* input, float* output, int edge) {

	KERNEL_POSITION;
	output[position] = (input[position] < 0 ? 0 : input[position]) + 1.3f;
}

template<>
__global__ void relu(halfloat* input, halfloat* output, int edge) {

	KERNEL_POSITION;

	halfloat zero = 0.0f;
	halfloat add = 1.3f;
	output[position] = (input[position] < zero ? zero : input[position]) + add;
}

nvinfer1::Dims WReLU::outputDims(int index, const nvinfer1::Dims* inputDims, int nbInputDims) {
	return inputDims[0];
}

std::shared_ptr<LayerConfig> WReLU::config(const std::string& layerName) {
	auto cfg = TRTPlugin::config(layerName);

	//��������������֧��half��float��ʽ
	cfg->supportDataType_ = {nvinfer1::DataType::kHALF, nvinfer1::DataType::kFLOAT};
	//cfg->supportDataType_ = {nvinfer1::DataType::kHALF};
	return cfg;
}

int WReLU::enqueue(const std::vector<Plugin::GTensor>& inputs, std::vector<Plugin::GTensor>& outputs, const std::vector<GTensor>& weights, void* workspace, hipStream_t stream) {

	int count = inputs[0].count();
	auto grid = gridDims(count);
	auto block = blockDims(count);

	if (config_->configDataType_ == TRTInfer::DataType::dtFloat) {
		relu <<<grid, block >>> (inputs[0].ptr<float>(), outputs[0].ptr<float>(), count);
	}

	//����������half������£�����half�ķ�����Ч�ʻ�Ƚϸߣ�����half2�����ת��Ϊhalf2�����и��ߵļ���
	else if (config_->configDataType_ == TRTInfer::DataType::dtHalfloat) {
		relu <<<grid, block>>> (inputs[0].ptr<halfloat>(), outputs[0].ptr<halfloat>(), count);
	}
	return 0;
}

RegisterPlugin(WReLU);